#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#include "Scene.h"
#include "Scene.cpp"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

void pathtraceInit(Scene* scene) {
	hst_scene = scene;
	const Camera& cam = hst_scene->state.cam;
	const int pixelcount = cam.resX * cam.resY;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_rays, pixelcount * sizeof(Ray));

	hipMalloc(&dev_objs, scene->sceneObjs.size() * sizeof(Object));
	hipMemcpy(dev_objs, scene->sceneObjs.data(), scene->sceneObjs.size() * sizeof(Object), hipMemcpyHostToDevice);

	hipMalloc(&dev_hits, pixelcount * sizeof(Hit));
	hipMemset(dev_hits, 0, pixelcount * sizeof(Hit));


}

void pathtraceFree() {
	hipFree(dev_image);  
	hipFree(dev_paths);
	hipFree(dev_objs);
	hipFree(dev_intersections);

}

__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, Ray* rays)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resX && y < cam.resY) {
		int index = x + (y * cam.resX);
		Ray& ray = rays[index];

		float sizeY = 2.f * cam.lensDistance * tan(cam.vFov);
		float sizeX = (float)cam.resX / (float)cam.resY * sizeY;
		float minX = (float)x / (float)cam.resX * sizeX - sizeX / 2.f;
		float maxX = (float)(x + 1) / (float)cam.resX * sizeX - sizeX / 2.f;
		float minY = (float)y / (float)cam.resY * sizeY - sizeY / 2.f;
		float maxY = (float)(y + 1) / (float)cam.resY * sizeY - sizeY / 2.f;
		float x = randf() * (maxX - minX) + minX; 
		float y = randf() * (maxY - minY) + minY; 
		float z = -cam.lensDistance; 
		
		Vec3 d = vecNormalize(Vec3(x, y, z));
		Vec3 o = Vec3(0.f);

		Transform vecTransform = cam.transform;
		vecTransform.pos = Vec3(0.f);
		ray.d = vecTransform.matVecMult(vecTransform.localToWorld(), d);
		ray.o = cam.transform.matVecMult(cam.transform.localToWorld(), o);

	}
}

__global__ void computeIntersections(
	int depth, int num_rays, Ray* rays, int objs_size, Objs* objs, Hit* hits
)
{
	int ray_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (ray_index < num_rays)
	{
		Ray ray = rays[ray_index];

		if (pathSegment.pixelIndex == 400) {
			int pixelIndex = pathSegment.pixelIndex;
			pixelIndex++;
		}

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_obj_index = -1;
		Hit& h;

		for (int i = 0; i < objs_size; i++)
		{
			Object& obj = objs[i];

			
			t = obj.hit(ray, h)
			
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
			}
		}

		if (hit_geom_index == -1)
		{
			hits[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			hits[ray_index].t = t_min;
			hits[ray_index].Mat = objs[hit_obj_index].Mat;
			hits[ray_index].surfaceNormal = h.normS;
		}
	}
}

void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resX * cam.resY;
	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resX + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resY + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_rays);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_ray_end = dev_rays + pixelcount;
	int num_rays = dev_ray_end - dev_rays;

	startCpuTimer();
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_hits, 0, pixelcount * sizeof(Hit));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_rays + blockSize1d - 1) / blockSize1d;

		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_rays
			, dev_rays
			, dev_objs
			, hst_scene->objs.size()
			, dev_hits
			);

		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;


		//printf("num paths: %i , depth: %i \n", num_paths, depth);
		if (num_rays == 0 || depth > traceDepth) {
			iterationComplete = true; // TODO: should be based off stream compaction results.
		}
	}
	num_rays = dev_ray_end - dev_rays;
	printf("Iteration Done\n");
	endCpuTimer();
	printTime();
	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	// finalGather << <numBlocksPixels, blockSize1d >> > (num_rays, dev_image, dev_rays);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->cam.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}