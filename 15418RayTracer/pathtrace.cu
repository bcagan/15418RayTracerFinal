#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

#include "Scene.h"
#include ""
#include "Intersections.h"
#include "Ray.h"
//#include "Ray.cpp"
#include "Transform.h"
//#include "Transform.cpp"
#include "Object.h"
//#include "Object.cpp"
//#include "Scene.cpp"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"


//https://stackoverflow.com/questions/6061565/setting-up-visual-studio-intellisense-for-cuda-kernel-calls
#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

#define cudaCheckError(ans)  cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s at %s:%d\n",
			hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

static Scene* hst_scene = NULL;
static Vec3* dev_image = NULL;
static Color3* dev_finalImage = NULL;
static Object* dev_objs = NULL;
static Material* dev_materials = NULL;
static Ray* dev_rays = NULL;
static Hit* dev_hits = NULL;
static int* dev_hitPeaks = NULL;
static int* dev_hitIndices = NULL;


/////////////////////////////
//Scan code from assignment 2
/////////////////////////////

static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void upsweepKernel(int N, int* data, int twod1, int twod) {
    // toWrite[0] = 51;
    // data[0] = 42;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = twod1 * index;
    //data[index] = data[i + twod1 -1] + data[i + twod -1] ;
    if (i < N) {
        int res = data[i + twod1 - 1] + data[i + twod - 1];
        data[i + twod1 - 1] = res;
    }
}

__global__ void downsweepKernel(int N, int* data, int twod1, int twod) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index * twod1;
    if (i < N) {
        int t = data[i + twod - 1] + data[i + twod1 - 1];
        int s = data[i + twod1 - 1];
        data[i + twod - 1] = s;
        data[i + twod1 - 1] = t;
    }
}

__global__ void set0(int N, int* deviceData) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == N - 1) deviceData[index] = 0;
    //else deviceData[index] = 1;
}

void exclusive_scan(int* device_data, int length) {
 
    int N = nextPow2(length);
    const int threadsPerBlock = 512;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    for (int twod = 1; twod < N; twod *= 2) {
        int twod1 = twod * 2;
        blocks = (N / twod1 + 1 + threadsPerBlock - 1) / threadsPerBlock;

        upsweepKernel CUDA_KERNEL(blocks, threadsPerBlock) (N, device_data, twod1, twod);
		//upsweepKernel<<<blocks,threadsPerBlock>>>(N, device_data, twod1, twod);
    }
    blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    set0 CUDA_KERNEL(blocks, threadsPerBlock) (N, device_data);

    for (int twod = N / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        blocks = (N / twod1 + 1 + threadsPerBlock - 1) / threadsPerBlock;
        downsweepKernel CUDA_KERNEL(blocks, threadsPerBlock) (N, device_data, twod1, twod);
    }

}



void cudaScan(int* inarray, int* end, int* resultarray) {
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    cudaCheckError(hipMalloc((void**)&device_data, sizeof(int) * rounded_length));

    cudaCheckError(hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice));

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost));
}



__global__ void intSet(int N, int* set, int to) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    set[index] = to;
}

__global__ void contractOut(int N, int* rays, int* indices, int* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N && rays[index] != 0) {
        out[indices[index]] = index;
    }
}

void debugAssist(int* from, int N) {
    //    printf("deb\n");
	int printArr[1000]; // int printArr[N];
    cudaCheckError(hipMemcpy(printArr, from, N * sizeof(int), hipMemcpyDeviceToHost));
    for (int c = 0; c < N; c++) {
        printf("%d ", printArr[c]);
    }
    printf("\n\n\n");
}

int concat_rays(int num_rays, int numblocksPathSegmentTracing, int blockSize1d, int* device_output) {

    //Peaks have been found in dev_hitPeaks

    //Get number of elements
    int* device_num;
	int rounded_length = nextPow2(num_rays);

    cudaCheckError(hipMalloc(&device_num, sizeof(int) * (rounded_length)));
	
    cudaScan(dev_hitPeaks, dev_hitPeaks + (num_rays + 1), device_num);
    int numberRays = 0;
    cudaCheckError(hipMemcpy(&numberRays, device_num + (num_rays), sizeof(int), hipMemcpyDeviceToHost));
	
    contractOut CUDA_KERNEL(numblocksPathSegmentTracing, blockSize1d) (num_rays, dev_hitPeaks, device_num, device_output);

    cudaCheckError(hipDeviceSynchronize());
	hipFree(device_num);

    return numberRays;

}

//New Path Tracer code

void pathtraceInit(Scene* scene) {
	hst_scene = scene;
	const Camera& cam = hst_scene->cam;
	const int pixelcount = cam.resX * cam.resY;

	hipMalloc(&dev_image, pixelcount * sizeof(Vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(Vec3));

	hipMalloc(&dev_finalImage, pixelcount * sizeof(Color3));
	hipMemset(dev_finalImage, 0, pixelcount * sizeof(Color3));

	hipMalloc(&dev_rays, pixelcount * sizeof(Ray));
	hipMemset(dev_rays, 0, pixelcount * sizeof(Ray));

	hipMalloc(&dev_objs, scene->sceneObjs.size() * sizeof(Object));
	hipMemcpy(dev_objs, scene->sceneObjs.data(), scene->sceneObjs.size() * sizeof(Object), hipMemcpyHostToDevice);

	hipMalloc(&dev_hits, pixelcount * sizeof(Hit));
	hipMemset(dev_hits, 0, pixelcount * sizeof(Hit));

	hipMalloc(&dev_hitPeaks, (pixelcount+1) * sizeof(int));
	hipMemset(dev_hitPeaks, 0, (pixelcount+1) * sizeof(int));

	hipMalloc(&dev_hitIndices, (pixelcount) * sizeof(int));
	hipMemset(dev_hitIndices, 0, (pixelcount) * sizeof(int));

}

void pathtraceFree() {
	hipFree(dev_image);  
	hipFree(dev_rays);
	hipFree(dev_objs);
	hipFree(dev_hits);
	hipFree(dev_hitPeaks);
	hipFree(dev_hitIndices);
}

__device__ Mat4x4 tmakeTransform(Transform* t) {
	//Create rotation matrices
	//Will do out just to make CUDA translation possible
	//Mat3x3 defined by each column 

	//Rotation
	Mat3x3 Rx = Mat3x3(Vec3(1.f, 0.f, 0.f), Vec3(0.f, cos(t->rot.x), sin(t->rot.x)), Vec3(0.f, -sin(t->rot.x), cos(t->rot.x)));
	Mat3x3 Ry = Mat3x3(Vec3(cos(t->rot.y), 0.f, -sin(t->rot.y)), Vec3(0.f, 1.f, 0.f), Vec3(sin(t->rot.y), 0.f, cos(t->rot.y)));
	Mat3x3 Rz = Mat3x3(Vec3(cos(t->rot.z), sin(t->rot.z), 0.f), Vec3(-sin(t->rot.z), cos(t->rot.z), 0.f), Vec3(0.f, 0.f, 1.f));
	Mat3x3 R = t->matMult(Rx, t->matMult(Ry, Rz));

	//Position
	Mat4x3 P = Mat4x3(Vec3(1.f, 0.f, 0.f), Vec3(0.f, 1.f, 0.f), Vec3(0.f, 0.f, 1.f), t->pos);

	//Scaling
	Mat3x3 S = Mat3x3(Vec3(t->scale.x, 0.f, 0.f), Vec3(0.f, t->scale.y, 0.f), Vec3(0.f, 0.f, t->scale.z));

	//Scale and rotate before position, scaling and rotation can be swapped 
	Mat3x3 RS = t->matMult(R, S);
	Mat4x4 preRes = Mat4x4(t->matMult(P, RS));
	preRes.set(3, 3, 1.f);
	t->tempMatrix = preRes;
}

__device__ Mat4x4 localToWorld(Transform t) {
	if (!t.tempMatrixFilled) tmakeTransform(&t);
	Mat4x4 res = t.tempMatrix; //So, take the local spa
	//printf("parent: %lu\n", (unsigned long) t.parent);
	Transform* tp = t.parent;
	while (tp != nullptr) {
		res = (t.parent)->tempMatrix * res;
		tp = t.parent;
	}
	/*if (t.parent != nullptr) 
	res = localToWorld(*(t.parent)) * res;*/
	return res;
}

__global__ void generateRayFromCamera(Camera cam, int traceDepth, Ray* rays, int seed)
{
	int ix = (blockIdx.x * blockDim.x) + threadIdx.x;
	int iy = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (ix < cam.resX && iy < cam.resY) {
		int index = ix + (iy * cam.resX);

		// printf("Hello from pixel %d\n", index);
		Ray& ray = rays[index];

		float sizeY = 2.f * cam.lensDistance * tan(cam.vFov);
		float sizeX = (float)cam.resX / (float)cam.resY * sizeY;
		float minX = (float)ix / (float)cam.resX * sizeX - sizeX / 2.f;
		float maxX = (float)(ix + 1) / (float)cam.resX * sizeX - sizeX / 2.f;
		float minY = (float)iy / (float)cam.resY * sizeY - sizeY / 2.f;
		float maxY = (float)(iy + 1) / (float)cam.resY * sizeY - sizeY / 2.f;

		hiprandState state;
		hiprand_init(12345 + seed * index, 0, 0, &state);

		float rand1 = hiprand_uniform_double(&state);
		float rand2 = hiprand_uniform_double(&state);


		float x = rand1 * (maxX - minX) + minX; 
		float y = rand2 * (maxY - minY) + minY; 
		float z = -cam.lensDistance; 

		
		Vec3 d = vecNormalize(Vec3(x, y, z));
		Vec3 o = Vec3(0.f);

		Transform vecTransform = cam.transform;
		vecTransform.pos = Vec3(0.f);

		ray.d = vecTransform.matVecMult(localToWorld(vecTransform), d);
		ray.o = cam.transform.matVecMult(localToWorld(cam.transform), o);
		ray.pixelIndex = index;
		ray.maxt = INFINITY;
		ray.mint = EPSILON;
		ray.numBounces = traceDepth;
		ray.color = Vec3(0);
		ray.storeColor = Vec3(1.f);

		//printf("pixX %d pixY %d minX maxX miny maxY %f %f %f %f x y %f %f \n", ix, iy, minX, maxX, minY, maxY, x, y);
		// if(index == 200) printf("xx y z %f %f %f \n", rays[index].maxt, rays[index].mint, rays[index].d.z);

	}
}

__device__ inline Vec3 rrandomOnUnitSphere(float cosphi, float theta) {

	float sinphi = sqrt(1.f - cosphi * cosphi);
	float x = cos(theta) * sinphi;
	float z = sin(theta) * sinphi;
	float y = cosphi;
	return Vec3(x, y, z);
}

__device__ Vec3 bounce(Hit* hits, int ray_index, int seed) {
	Hit& h = hits[ray_index];
	hiprandState state;
	hiprand_init(4321 + seed * ray_index, 0, 0, &state);

	float rand1 = hiprand_uniform_double(&state);
	float rand2 = hiprand_uniform_double(&state);

	float theta = 2.f * rand1 * PI;
	float cosphi = 2.f * rand2 - 1.f;

	return vecNormalize(vecVecAdd(h.normS, rrandomOnUnitSphere(cosphi, theta)));
}

__global__ void calculateColor(Camera cam, Ray* rays, Hit* hits, int iter, int num_rays, int seed)
{
	int ray_index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (ray_index < num_rays) {
		
		Ray& r = rays[ray_index];
		Hit& hit = hits[ray_index];

		if (hit.t != -1.0f) {
			// calculate bounce ray
			/*if (ray_index == 500) printf("oldr origin x: %f y: %f z: %f  \n", r.o.x, r.o.y, r.o.z);
			if (ray_index == 500) printf("oldr direction x: %f y: %f z: %f  \n", r.d.x, r.d.y, r.d.z);*/
			Vec3 bouncedHit = bounce(hits, ray_index, seed);
			
			Ray newR = Ray(vecVecAdd(constVecMult(hit.t, r.d), r.o), bouncedHit);
			/*if (ray_index == 500) printf("newrr origin x: %f y: %f z: %f  \n", newR.o.x, newR.o.y, newR.o.z);
			if (ray_index == 500) printf("newr direction x: %f y: %f z: %f  \n", newR.d.x, newR.d.y, newR.d.z);*/
		
			newR.color = Color3(hit.emitted().toVec3() * r.storeColor + r.color).toVec3();
			newR.storeColor = Color3(r.storeColor * hit.albedo().toVec3()).toVec3();

			if (ray_index == 5000) {
				/*printf("old vs newRGB r: %f, g: %f, b: %f  r: %f, g: %f, b: %f \n", r.color.r, r.color.g, r.color.b, newR.color.r, newR.color.g, newR.color.b);
				printf("newR.storeColor r: %u %u %u alb: %f %f %f\n ", newR.storeColor.r, newR.storeColor.g, newR.storeColor.b, hit.albedo().toVec3().x, hit.albedo().toVec3().y, hit.albedo().toVec3().z);*/
			}

			// set up for next bounce 
			r.d = newR.d;
			r.o = newR.o;
			r.mint = EPSILON;
			r.maxt = INFINITY;
			r.color = newR.color;
			r.storeColor = newR.storeColor;
			r.numBounces--;
			

			if (ray_index == 5000) {
				//printf("rgb r: %f, g: %f, b: %f bounce: %f\n", r.color.r, r.color.g, r.color.b, r.numBounces);
			}

		}
		

	}
}

__device__ double stdmin(double a, double b) {
	if (a > b) return b;
	else return a;
}

__device__ double stdmax(double a, double b) {
	if (a > b) return a;
	else return b;
}

//__device__ bool bboxHit(Object& o, Ray& r, Hit& hit, int ray_index) {
__device__ bool bboxHit(int obj_index, int ray_index, Ray* rays, Object* objs, Hit* hits) {
	
	BBox& b = objs[obj_index].bbox;
	Vec3 min = b.min;
	Vec3 max = b.max;
	
	Ray& r = rays[ray_index];
	Hit& hit = hits[ray_index];
	

	double tmin = -INFINITY, tmax = INFINITY;

	Vec3 invdir = vecVecDiv(Vec3(1.f), r.d);

	// value of t in the parametric ray equation where ray intersects min coordinate with dimension i
	double t1 = (min.x - r.o.x) * invdir.x;
	// value of t in the parametric ray equation where ray intersects max coordinate with dimension i
	double t2 = (max.x - r.o.x) * invdir.x;

	tmin = stdmax(tmin, stdmin(t1, t2));
	tmax = stdmin(tmax, stdmax(t1, t2));

	t1 = (min.y - r.o.y) * invdir.y;
	t2 = (max.y - r.o.y) * invdir.y;

	tmin = stdmax(tmin, stdmin(t1, t2));
	tmax = stdmin(tmax, stdmax(t1, t2));

	t1 = (min.z - r.o.z) * invdir.z;
	t2 = (max.z - r.o.z) * invdir.z;

	tmin = stdmax(tmin, stdmin(t1, t2));
	tmax = stdmin(tmax, stdmax(t1, t2));

	/*if (ray_index == 500) printf("DIR rx: %f ry: %f rz: %f bboxx: %f bboxy: %f bboxz: %f\n", r.d.x, r.d.y, r.d.z, min.x, min.y, min.z);
	if (ray_index == 500) printf("OUTSIDE tmin: %f tmax: %f hit.t: %f r.maxt: %f \n", tmin, tmax, hit.t, r.maxt);*/

	if (r.maxt >= tmin && tmax >= tmin && tmin > EPSILON) {
		
		hit.t = tmin;
		
		Vec3 pos = vecVecDiv(vecVecAdd(b.max, b.min), Vec3(2.0f));
		hit.uv = Vec2(0.f);
		/*if (ray_index == 500) printf("tmin: %f tmax: %f hit.t: %f r.maxt: %f \n", tmin, tmax, hit.t, r.maxt);*/
		return true;
	}
	return false;
}

__device__ bool cubeHit(int obj_index, int ray_index, Ray* rays, Object* objs, Hit* hits) {
	Object& o = objs[obj_index];
	Ray& ray = rays[ray_index];
	Hit& hit = hits[ray_index];
	//Hit temp;
	if (hit.t < ray.maxt) {
		hit.Mat = o.Mat;
		
		Vec3 normVec = vecNormalize(vecVecAdd((vecVecAdd(ray.o, constVecMult(hit.t, ray.d))), constVecMult(-1.f, o.pos)));
		/*if (ray_index == 500) printf("NORMVEC: %f %f %f \n", normVec.x, normVec.y, normVec.z);*/
		if (abs(normVec.x) > abs(normVec.y) && abs(normVec.x) > abs(normVec.z)) {
			if (normVec.x < 0) hit.normG = Vec3(-1.f, 0.f, 0.f);
			else hit.normG = Vec3(1.f, 0.f, 0.f);
		}
		else if (abs(normVec.y) > abs(normVec.x) && abs(normVec.y) > abs(normVec.z)) {
			if (normVec.y < 0) hit.normG = Vec3(0.f, -1.f, 0.f);
			else hit.normG = Vec3(0.f, 1.f, 0.f);
		}
		else {
			if (normVec.z < 0) hit.normG = Vec3(0.f, 0.f, -1.f);
			else hit.normG = Vec3(0.f, 0.f, 1.f);
		}
		hit.normS = hit.normG;
		hit.uv = Vec2(0.f);//Not doing right now
		ray.maxt = hit.t;
		return true;
	}

	return false;
}

__device__ void swap(float a, float b) {
	float c(a); a = b; b = c;
}

__device__ bool sphereHit(int obj_index, int ray_index, Ray* rays, Object* objs, Hit* hits) {
	Object& o = objs[obj_index];
	Ray& r = rays[ray_index];
	Hit& h = hits[ray_index];

	float t0, t1;
	const Vec3 L = vecVecAdd(vecVecSub(o.pos, r.o), constVecMult(r.mint, r.d));
	const double tca = dot(L, r.d);
	// ignore if vector is facing the opposite way in any direction
	if (tca < 0) return false;
	const double d2 = dot(L, L) - tca * tca;
	const double radius2 = o.sz * o.sz;
	if (d2 > radius2) return false;
	const double thc = sqrt(radius2 - d2);
	t0 = tca - thc;
	t1 = tca + thc;

	if (t0 > t1) swap(t0, t1);

	if (t0 < 0) {
		t0 = t1; // if t0 is negative, let's use t1 instead 
		if (t0 < 0) return false; // both t0 and t1 are negative 
	}

	if (h.t >= t0 && t0 > r.mint) {
		if (t0 > r.mint) {
			h.t = t0;
		}
		else {
			h.t = r.mint;
		}
		h.normG = vecNormalize(vecVecSub(vecVecAdd(r.o, constVecMult(h.t, r.d)), o.pos));
		h.normS = h.normG;
		h.uv = Vec2(0.f);
	}
	return true;
}

__global__ void computeIntersections(
	int depth, int num_rays, Ray* rays, int objs_size, Object* objs, Hit* hits, int* hitPeaks, int* hitIndices
)
{
	int ray_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (ray_index < num_rays)
	{
		bool hitbool = false;
		Ray& ray = rays[ray_index];
		int hit_obj_index = -1;
		Hit& h = hits[ray_index];

		for (int i = 0; i < objs_size; i++)
		{
			Object& obj = objs[i];

			
			
			if (bboxHit(i, ray_index, rays, objs, hits)) {
				
				if (obj.type == gcube) {
					if (cubeHit(i, ray_index, rays, objs, hits)) {
						if (h.t < ray.maxt) {
							ray.maxt = h.t;
							h.Mat = obj.Mat;
						}
						
						hitbool = true;
					}
				}
				else if (obj.type == gsphere) {
					if (sphereHit(i, ray_index, rays, objs, hits)) {
						h.Mat = obj.Mat;
					}
				}		
			}
			
		}

		if (!hitbool)
		{
			h.t = -1.0f;
			hitPeaks[ray_index + 1] = 0;
		}
		else
		{
			
			hitPeaks[ray_index + 1] = 1;
			
		}

		
	}
}

__global__ void fillIndices(int num_rays, int* hitIndices) {
	int indexIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (indexIndex < num_rays) {
		hitIndices[indexIndex] = indexIndex;
	}
}

__global__ void finalGather(int num_rays, Vec3* image, Ray* rays, int num_samples)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < num_rays)
	{
		Ray ray = rays[index];
		image[ray.pixelIndex] += (ray.color /= (float)num_samples);
	}
}

__global__ void color3Gather(int num_rays, Vec3* image, Color3* newImage)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < num_rays)
	{
		newImage[index] = Color3(image[index]);
	}
}

void pathtrace(int iter) {
	// it might make more sense to define number of ray bounces in the scene rather than the ray
	const int traceDepth = iter;
	const Camera& cam = hst_scene->cam;
	const int pixelcount = cam.resX * cam.resY;
	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resX + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resY + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;
	int numblocksPathSegmentTracing;
	
	//Pre define final block size for image storage
	dim3 numBlocksPixels;
	

	//
	
	for (int s = 0; s < hst_scene->sampleNum(); s++) {

		int seed = (int)(rand() * 100);
		printf("seed %d \n", seed);
		generateRayFromCamera CUDA_KERNEL(blocksPerGrid2d, blockSize2d) (cam, traceDepth, dev_rays, seed);

		int depth = 0;
		Ray* dev_ray_end = dev_rays + pixelcount;
		int num_rays = dev_ray_end - dev_rays;

		// --- PathSegment Tracing Stage ---
		// Shoot ray into scene, bounce between objects, push shading chunks
		bool iterationComplete = false;
		while (!iterationComplete) {
			
			// clean shading chunks
			hipMemset(dev_hits, 0, pixelcount * sizeof(Hit));

			// tracing
			numblocksPathSegmentTracing = (num_rays + blockSize1d - 1) / blockSize1d;

			fillIndices CUDA_KERNEL(numblocksPathSegmentTracing, blockSize1d) (num_rays, dev_hitIndices); //Fill init indices to 1...num_rays

			computeIntersections CUDA_KERNEL(numblocksPathSegmentTracing, blockSize1d) (
				depth
				, num_rays
				, dev_rays
				, hst_scene->sceneObjs.size()
				, dev_objs
				, dev_hits
				, dev_hitPeaks
				, dev_hitIndices
				);

			seed = (int)(rand() * 100);


			calculateColor CUDA_KERNEL(numblocksPathSegmentTracing, blockSize1d) (cam, dev_rays, dev_hits, depth, num_rays, seed);

			hipDeviceSynchronize();
			depth++;

			//Use find rays to contract rays into those that have ended and those that havent

			//num_rays = concat_rays(num_rays, numblocksPathSegmentTracing, blockSize1d, dev_hitIndices);

			printf("num rays: %i , depth: %i, tracedepth: %i \n", num_rays, depth, traceDepth);
			if (num_rays == 0 || depth > traceDepth) {
				iterationComplete = true; // TODO: should be based off stream compaction results.
			}
		}
		num_rays = dev_ray_end - dev_rays;
		//printf("gathered %d\n", dev_rays[2000].color);
		printf("Iteration Done\n");

		// Assemble this iteration and apply it to the image
		numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
		finalGather CUDA_KERNEL(numBlocksPixels, blockSize1d) (num_rays, dev_image, dev_rays, hst_scene->sampleNum());

	}

	color3Gather CUDA_KERNEL(numBlocksPixels,blockSize1d)(pixelcount,dev_image,dev_finalImage);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	// sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->cam.img.data(), dev_finalImage,
		pixelcount * sizeof(Color3), hipMemcpyDeviceToHost);

}