#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <cmath>

#include "Scene.h"
#include "Scene.cpp"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Object* dev_objs = NULL;
static Material* dev_materials = NULL;
static Ray* dev_rays = NULL;
static Hit* dev_hits = NULL;
static int* dev_hitPeaks = NULL;
static int* dev_hitIndices = NULL;


/////////////////////////////
//Scan code from assignment 2
/////////////////////////////

static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void upsweepKernel(int N, int* data, int twod1, int twod) {
    // toWrite[0] = 51;
    // data[0] = 42;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = twod1 * index;
    //data[index] = data[i + twod1 -1] + data[i + twod -1] ;
    if (i < N) {
        int res = data[i + twod1 - 1] + data[i + twod - 1];
        data[i + twod1 - 1] = res;
    }
}

__global__ void downsweepKernel(int N, int* data, int twod1, int twod) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index * twod1;
    if (i < N) {
        int t = data[i + twod - 1] + data[i + twod1 - 1];
        int s = data[i + twod1 - 1];
        data[i + twod - 1] = s;
        data[i + twod1 - 1] = t;
    }
}

__global__ void set0(int N, int* deviceData) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == N - 1) deviceData[index] = 0;
    //else deviceData[index] = 1;
}

void exclusive_scan(int* device_data, int length) {
 
    int N = nextPow2(length);
    int printArr[N];
    const int threadsPerBlock = 512;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    for (int twod = 1; twod < N; twod *= 2) {
        int twod1 = twod * 2;
        blocks = (N / twod1 + 1 + threadsPerBlock - 1) / threadsPerBlock;

        upsweepKernel << <blocks, threadsPerBlock >> > (N, device_data, twod1, twod);
    }
    blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    set0 << <blocks, threadsPerBlock >> > (N, device_data);

    for (int twod = N / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        blocks = (N / twod1 + 1 + threadsPerBlock - 1) / threadsPerBlock;
        downsweepKernel << <blocks, threadsPerBlock >> > (N, device_data, twod1, twod);
    }

}



double cudaScan(int* inarray, int* end, int* resultarray) {
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    cudaCheckError(hipMalloc((void**)&device_data, sizeof(int) * rounded_length));

    cudaCheckError(hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice));

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost));
}



__global__ void intSet(int N, int* set, int to) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    set[index] = to;
}

__global__ void contractOut(int N, int* rays, int* indices, int* out, int x, int y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N && rays[index] != 0) {
        out[indices[index]] = index;
    }
}

void debugAssist(int* from, int N) {
    //    printf("deb\n");
    int printArr[N];
    cudaCheckError(hipMemcpy(printArr, from, N * sizeof(int), hipMemcpyDeviceToHost));
    for (int c = 0; c < N; c++) {
        printf("%d ", printArr[c]);
    }
    printf("\n\n\n");
}

int concat_rays(int num_rays, int numblocksPathSegmentTracing, int blockSize1d, int* device_output) {
    

	//int indexIndex = blockIdx.x * blockDim.x + threadIdx.x;
	//int ray_index = hitIndices[indexIndex];

    //Peaks have been found in dev_hitPeaks

    //Get number of elements
    int* device_num;
    cudaCheckError(hipMalloc(&device_num, sizeof(int) * (num_rays + 1)));
    cudaScan(dev_hitPeaks, dev_hitPeaks + (num_rays + 1), device_num);
    int numberRays = 0;
    cudaCheckError(hipMemcpy(&numberRays, device_num + (num_rays), sizeof(int), hipMemcpyDeviceToHost));

    contractOut <<<numblocksPathSegmentTracing, blockSize1d >> > (num_rays, dev_hitPeaks, device_num, device_output, x, y);

    cudaCheckError(hipDeviceSynchronize());
	hipFree(device_num);

    return numberRays;

}

//New Path Tracer code

void pathtraceInit(Scene* scene) {
	hst_scene = scene;
	const Camera& cam = hst_scene->cam;
	const int pixelcount = cam.resX * cam.resY;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_rays, pixelcount * sizeof(Ray));

	hipMalloc(&dev_objs, scene->sceneObjs.size() * sizeof(Object));
	hipMemcpy(dev_objs, scene->sceneObjs.data(), scene->sceneObjs.size() * sizeof(Object), hipMemcpyHostToDevice);

	hipMalloc(&dev_hits, pixelcount * sizeof(Hit));
	hipMemset(dev_hits, 0, pixelcount * sizeof(Hit));

	hipMalloc(&dev_hitPeaks, (pixelcount+1) * sizeof(int));
	hipMemset(dev_hitPeaks, 0, (pixelcount+1) * sizeof(int));

	hipMalloc(&dev_hitIndices, (pixelcount) * sizeof(int));
	hipMemset(dev_hitIndices, 0, (pixelcount) * sizeof(int));


}

void pathtraceFree() {
	hipFree(dev_image);  
	hipFree(dev_rays);
	hipFree(dev_objs);
	hipFree(dev_hits);
	hipFree(dev_blockHits);
	hipFree(dev_hitIndices);
}

__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, Ray* rays)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resX && y < cam.resY) {
		int index = x + (y * cam.resX);
		Ray& ray = rays[index];

		float sizeY = 2.f * cam.lensDistance * tan(cam.vFov);
		float sizeX = (float)cam.resX / (float)cam.resY * sizeY;
		float minX = (float)x / (float)cam.resX * sizeX - sizeX / 2.f;
		float maxX = (float)(x + 1) / (float)cam.resX * sizeX - sizeX / 2.f;
		float minY = (float)y / (float)cam.resY * sizeY - sizeY / 2.f;
		float maxY = (float)(y + 1) / (float)cam.resY * sizeY - sizeY / 2.f;
		float x = randf() * (maxX - minX) + minX; 
		float y = randf() * (maxY - minY) + minY; 
		float z = -cam.lensDistance; 
		
		Vec3 d = vecNormalize(Vec3(x, y, z));
		Vec3 o = Vec3(0.f);

		Transform vecTransform = cam.transform;
		vecTransform.pos = Vec3(0.f);
		ray.d = vecTransform.matVecMult(vecTransform.localToWorld(), d);
		ray.o = cam.transform.matVecMult(cam.transform.localToWorld(), o);

	}
}


__global__ void computeIntersections(
	int depth, int num_rays, Ray* rays, int objs_size, Object* objs, Hit* hits, int* hitPeaks, int* hitIndices
)
{
	int indexIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int ray_index = hitIndices[indexIndex];

	if (indexIndex < num_rays)
	{
		Ray ray = rays[ray_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_obj_index = -1;
		Hit h;

		for (int i = 0; i < objs_size; i++)
		{
			Object& obj = objs[i];

			
			t = obj.hit(ray, h);
			
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_obj_index = i;
			}
		}
		
		if (hit_obj_index == -1)
		{
			hits[ray_index].t = -1.0f;
			hitPeaks[ray_index + 1] = 0;
		}
		else
		{
			//The ray hits something
			hits[ray_index].t = t_min;
			hits[ray_index].Mat = objs[hit_obj_index].Mat;
			hits[ray_index].normS = h.normS;
			hitPeaks[ray_index + 1] = 1;
		}

		
	}
}

__global__ void fillIndices(int num_rays, int* hitIndices) {
	int indexIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (indexIndex < num_rays) {
		hitIndices[indexIndex] = indexIndex;
	}
}

void pathtrace(int frame, int iter) {
	// it might make more sense to define number of ray bounces in the scene rather than the ray
	const int traceDepth = 15;
	const Camera& cam = hst_scene->cam;
	const int pixelcount = cam.resX * cam.resY;
	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resX + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resY + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_rays);

	int depth = 0;
	Ray* dev_ray_end = dev_rays + pixelcount;
	int num_rays = dev_ray_end - dev_rays;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_hits, 0, pixelcount * sizeof(Hit));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_rays + blockSize1d - 1) / blockSize1d;

		fillIndices << <numblocksPathSegmentTracing, blockSize1d >> > (num_rays, dev_hitIndices); //Fill init indices to 1...num_rays

		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_rays
			, dev_rays
			, dev_objs
			, hst_scene->objs.size()
			, dev_hits
			, dev_hitPeaks
			, dev_hitIndices
			);

		
		hipDeviceSynchronize();
		depth++;

		//Use find rays to contract rays into those that have ended and those that havent

		num_rays = concat_rays(num_rays, numblocksPathSegmentTracing, blockSize1d, dev_hitIndices) {

		//printf("num paths: %i , depth: %i \n", num_paths, depth);
		if (num_rays == 0 || depth > traceDepth) {
			iterationComplete = true; // TODO: should be based off stream compaction results.
		}
	}
	num_rays = dev_ray_end - dev_rays;
	printf("Iteration Done\n");
	
	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	// finalGather << <numBlocksPixels, blockSize1d >> > (num_rays, dev_image, dev_rays);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	// sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->cam.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

}